#include "hip/hip_runtime.h"
#include "Math/Vector4D.h"
#include "ROOT/RDataFrame.hxx"
#include "ROOT/RVec.hxx"
#include "TCanvas.h"
#include "device.h"
#include "flattened_jagged_vec.h"
#include "util.h"
#include "Histogram.h"
#include <TFile.h>
#include <TTree.h>
#include <iostream>
#include <vector>

class AnalysisWorkflow {
public:
  AnalysisWorkflow(const std::string &filename);
  ~AnalysisWorkflow();
  void Run();

private:
  // host attributes
  std::string filename_;
  std::vector<UInt_t> nJets;
  JaggedVec<float> Jet_pts;
  JaggedVec<float> Jet_etas;
  JaggedVec<float> Jet_phis;
  JaggedVec<float> Jet_masses;

  // transformed attributes to be used on the GPU
  // FlattendJaggedVec contains host and device members
  FlattenedJaggedVec<float> flattened_Jet_pts;
  FlattenedJaggedVec<float> flattened_Jet_etas;
  FlattenedJaggedVec<float> flattened_Jet_phis;
  FlattenedJaggedVec<float> flattened_Jet_masses;

  // device attribute without transformation
  UInt_t *device_nJets = nullptr;
  float *device_trijet_pt_bins = nullptr;

  // attribute only needed on the GPU -> no host equivalent
  FlattenedJaggedVec<DeviceXYZTVector> device_Jet_xyzts;

  int num_threads_per_block_ = 128;
  int num_histogram_bins_ = 100;
  Histogram trijet_pt_histogram_;

  void LoadAndFilterData();
  void FlattenJaggedAttributes();
  void CopyToDevice();
  void RunAnalysis();
  void CopyToHost();
  void GeneratePlots();
};

AnalysisWorkflow::AnalysisWorkflow(const std::string &filename)
    : filename_(filename),
      trijet_pt_histogram_("", ";Trijet pt (GeV);N_{Events}", /*[>nbins*/ 100, /*xin<]*/ 15,
          /*xmax*/ 40) {}

AnalysisWorkflow::~AnalysisWorkflow() {
  if (device_nJets) {
    hipFree(device_nJets);
    device_nJets = nullptr;
  }
}

void AnalysisWorkflow::Run() {
  LoadAndFilterData();
  FlattenJaggedAttributes();
  CopyToDevice();
  RunAnalysis();
  CopyToHost();
  GeneratePlots();
}

void AnalysisWorkflow::LoadAndFilterData() {
  //ROOT::EnableImplicitMT(); // Optional: Enable multi-threading
  std::string treename = "Events";
  ROOT::RDataFrame df(treename, filename_);
  auto df2 = df.Filter([](unsigned int n) { return n >= 3; }, {"nJet"},
                       "At least three jets");
  nJets = df2.Take<UInt_t>("nJet").GetValue();
  Jet_pts = df2.Take<ROOT::RVec<Float_t>>("Jet_pt").GetValue();
  Jet_etas = df2.Take<ROOT::RVec<Float_t>>("Jet_eta").GetValue();
  Jet_phis = df2.Take<ROOT::RVec<Float_t>>("Jet_phi").GetValue();
  Jet_masses = df2.Take<ROOT::RVec<Float_t>>("Jet_mass").GetValue();

}

void AnalysisWorkflow::FlattenJaggedAttributes() {
  flattened_Jet_pts = Jet_pts;
  flattened_Jet_etas = Jet_etas;
  flattened_Jet_phis = Jet_phis;
  flattened_Jet_masses = Jet_masses;
}

void AnalysisWorkflow::CopyToDevice() {
  cudaMallocAndCopy(nJets, &device_nJets);

  flattened_Jet_pts.CopyToDevice();
  flattened_Jet_etas.CopyToDevice();
  flattened_Jet_phis.CopyToDevice();
  flattened_Jet_masses.CopyToDevice();

  device_Jet_xyzts.ReserveDataAndCopySizesAndOffsetsToDevice(flattened_Jet_pts);

  trijet_pt_histogram_.CopyToDevice();
}

// For debugging purposes
__global__ void PrintBins(Histogram::DeviceHistogram trijet_pt_histogram) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= 1) {
    return;
  }
  for (int i = 0; i < trijet_pt_histogram.num_bins_; ++i) {
    printf("Bin %d: %f\n", i, trijet_pt_histogram.bins_[i]);
  }
}

void AnalysisWorkflow::RunAnalysis() {
  int num_blocks =
      (nJets.size() + num_threads_per_block_ - 1) / num_threads_per_block_;
  std::cout << "Num jets: " << nJets.size() << std::endl;
  std::cout << "Num threads: " << num_threads_per_block_ << std::endl;
  std::cout << "Num blocks: " << num_blocks << std::endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
  std::cout << "Max block dim (x, y, z): " 
            << prop.maxThreadsDim[0] << ", " 
            << prop.maxThreadsDim[1] << ", " 
            << prop.maxThreadsDim[2] << "\n";
  std::cout << "Max grid dim (x, y, z): " 
            << prop.maxGridSize[0] << ", " 
            << prop.maxGridSize[1] << ", " 
            << prop.maxGridSize[2] << "\n";

  AnalysisKernel<<<num_blocks, num_threads_per_block_>>>(
      nJets.size(), device_nJets, flattened_Jet_pts.GetDeviceAttr(),
      flattened_Jet_etas.GetDeviceAttr(), flattened_Jet_phis.GetDeviceAttr(),
      flattened_Jet_masses.GetDeviceAttr(), device_Jet_xyzts.GetDeviceAttr(),
      trijet_pt_histogram_.GetDeviceHistogram());
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::stringstream s;
    s << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    throw std::runtime_error(s.str());
  }
  PrintBins<<<1, 1>>>(trijet_pt_histogram_.GetDeviceHistogram());
}

void AnalysisWorkflow::CopyToHost() {
  // copy histogram bins back to host
  trijet_pt_histogram_.CopyToHost();
}

void AnalysisWorkflow::GeneratePlots() {
  TCanvas c;
  // c.Divide(2, 1);
  // c.cd(1);
  trijet_pt_histogram_.GetTH1F().Draw();
  // c.cd(2);
  // h2->Draw();
  c.SaveAs("6_rdataframe_compiled_gpu.pdf");
}

int main() {
  AnalysisWorkflow workflow("../../../data/Run2012B_SingleMu.root");
  workflow.Run();
  return 0;
}
