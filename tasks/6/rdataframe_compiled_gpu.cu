#include "hip/hip_runtime.h"
#include "DeviceLorentzVector.h"
#include "DevicePtEtaPhiMVector.h"
#include "DevicePxPyPzE4D.h"
#include "Math/Vector4D.h"
#include "ROOT/RDataFrame.hxx"
#include "ROOT/RVec.hxx"
#include "TCanvas.h"
#include "TH1F.h"
#include "flattened_jagged_vec.h"
#include "util.h"
#include <TFile.h>
#include <TTree.h>
#include <iostream>
#include <vector>

typedef DeviceLorentzVector<DevicePxPyPzE4D<double>> DeviceXYZTVector;

__global__ void AnalysisKernel();

class AnalysisWorkflow {
public:
  AnalysisWorkflow(const std::string &filename);
  ~AnalysisWorkflow();
  void Run();

private:
  // host attributes
  std::string filename_;
  std::vector<UInt_t> nJets;
  JaggedVec<float> Jet_pts;
  JaggedVec<float> Jet_etas;
  JaggedVec<float> Jet_phis;
  JaggedVec<float> Jet_masses;
  std::vector<float> host_trijet_pt_bins_;

  // tranformed attributes to be used on the GPU
  // FlattendJaggedVec contains host and device members
  FlattenedJaggedVec<float> flattened_Jet_pts;
  FlattenedJaggedVec<float> flattened_Jet_etas;
  FlattenedJaggedVec<float> flattened_Jet_phis;
  FlattenedJaggedVec<float> flattened_Jet_masses;

  // device attribute without transformation
  UInt_t *device_nJets = nullptr;
  float *device_trijet_pt_bins = nullptr;

  // attribute only needed on the GPU -> no host equivalent
  FlattenedJaggedVec<DeviceXYZTVector> device_Jet_xyzts;

  int num_threads_per_block_ = 128;
  int num_histogram_bins_ = 100;

  void LoadAndFilterData();
  void FlattenJaggedAttributes();
  void CopyToDevice();
  void RunAnalysis();
  void CopyToHost();
  void GeneratePlots();
};

AnalysisWorkflow::AnalysisWorkflow(const std::string &filename)
    : filename_(filename) {}

AnalysisWorkflow::~AnalysisWorkflow() {
  if (device_nJets) {
    hipFree(device_nJets);
    device_nJets = nullptr;
  }
}

void AnalysisWorkflow::Run() {
  LoadAndFilterData();
  FlattenJaggedAttributes();
  CopyToDevice();
  RunAnalysis();
  CopyToHost();
  GeneratePlots();
}

void AnalysisWorkflow::LoadAndFilterData() {
  ROOT::EnableImplicitMT(); // Optional: Enable multi-threading
  std::string treename = "Events";
  ROOT::RDataFrame df(treename, filename_);
  auto df2 = df.Filter([](unsigned int n) { return n >= 3; }, {"nJet"},
                       "At least three jets");
  nJets = df2.Take<UInt_t>("nJet").GetValue();
  Jet_pts = df2.Take<ROOT::RVec<Float_t>>("Jet_pt").GetValue();
  Jet_etas = df2.Take<ROOT::RVec<Float_t>>("Jet_eta").GetValue();
  Jet_phis = df2.Take<ROOT::RVec<Float_t>>("Jet_phi").GetValue();
  Jet_masses = df2.Take<ROOT::RVec<Float_t>>("Jet_mass").GetValue();

  // initialize histrogram bins
  host_trijet_pt_bins_.resize(num_histogram_bins_ + 2); // +2 for over/underflow
}

void AnalysisWorkflow::FlattenJaggedAttributes() {
  flattened_Jet_pts = Jet_pts;
  flattened_Jet_etas = Jet_etas;
  flattened_Jet_phis = Jet_phis;
  flattened_Jet_masses = Jet_masses;
}

void AnalysisWorkflow::CopyToDevice() {
  cudaMallocAndCopy(nJets, device_nJets);
  cudaMallocAndCopy(host_trijet_pt_bins_, device_trijet_pt_bins);

  flattened_Jet_pts.CopyToDevice();
  flattened_Jet_etas.CopyToDevice();
  flattened_Jet_phis.CopyToDevice();
  flattened_Jet_masses.CopyToDevice();

  device_Jet_xyzts.ReserveDataAndCopySizesAndOffsetsToDevice(flattened_Jet_pts);
}

void AnalysisWorkflow::RunAnalysis() {
  int num_blocks =
      (nJets.size() + num_threads_per_block_ - 1) / num_threads_per_block_;
  AnalysisKernel<<<num_threads_per_block_, 1>>>();
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::stringstream s;
    s << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    throw std::runtime_error(s.str());
  }
}

void AnalysisWorkflow::CopyToHost() {
  // copy histogram bins back to host
  cudaMallocAndCopy(host_trijet_pt_bins_, device_trijet_pt_bins);
}

void AnalysisWorkflow::GeneratePlots() {
  TH1F h1("", ";Trijet pt (GeV);N_{Events}", /*nbins*/ 100, /*xin*/ 15,
          /*xmax*/ 40);
  // int nbins = binContents.size();
  // double xmin = 0.0;
  // double xmax = static_cast<double>(nbins);

  // Set bin contents (ROOT bins are 1-indexed!)
  for (int i = 0; i < h1.GetNbinsX(); ++i) {
    h1.SetBinContent(i, host_trijet_pt_bins_[i]);
  }
  TCanvas c;
  // c.Divide(2, 1);
  // c.cd(1);
  h1.Draw();
  // c.cd(2);
  // h2->Draw();
  c.SaveAs("6_rdataframe_compiled.png");
}

__global__ void AnalysisKernel() {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // if (idx >= aw.nJets.size()) {
  // return;
  //}

  // auto JetXYZT = Construct<XYZTVector>(Construct<PtEtaPhiMVector>(pt, eta,
  // phi, m));}, Trijet_idx = find_trijet(JetXYZT); Trijet_pt = trijet_pt(pt,
  // eta, phi, m, Trijet_idx);
  //  histogram
  // atomicAdd(&trijet_pt_bins[bin_idx], 1);
}

#ifdef OFF
template <typename T> using Vec = const ROOT::RVec<T> &;
using ROOT::Math::XYZTVector;

XYZTVector typedef LorentzVector<PxPyPzE4D<double>> XYZTVector;
Construct PtEtaPhiMVector operator+ pt() Construct

    __device__ ROOT::RVec<std::size_t> find_trijet(Vec<XYZTVector> jets) {
  constexpr std::size_t n = 3;
  float distance = 1e9;
  const auto top_mass = 172.5;
  std::size_t idx1 = 0, idx2 = 1, idx3 = 2;

  for (std::size_t i = 0; i <= jets.size() - n; i++) {
    auto p1 = jets[i];
    for (std::size_t j = i + 1; j <= jets.size() - n + 1; j++) {
      auto p2 = jets[j];
      for (std::size_t k = j + 1; k <= jets.size() - n + 2; k++) {
        auto p3 = jets[k];
        const auto tmp_mass = (p1 + p2 + p3).mass();
        const auto tmp_distance = std::abs(tmp_mass - top_mass);
        if (tmp_distance < distance) {
          distance = tmp_distance;
          idx1 = i;
          idx2 = j;
          idx3 = k;
        }
      }
    }
  }
  return {idx1, idx2, idx3};
}

__device__ float trijet_pt(Vec<float> pt, Vec<float> eta, Vec<float> phi,
                           Vec<float> mass, Vec<std::size_t> idx) {
  auto p1 = ROOT::Math::PtEtaPhiMVector(pt[idx[0]], eta[idx[0]], phi[idx[0]],
                                        mass[idx[0]]);
  auto p2 = ROOT::Math::PtEtaPhiMVector(pt[idx[1]], eta[idx[1]], phi[idx[1]],
                                        mass[idx[1]]);
  auto p3 = ROOT::Math::PtEtaPhiMVector(pt[idx[2]], eta[idx[2]], phi[idx[2]],
                                        mass[idx[2]]);
  return (p1 + p2 + p3).pt();
}
#endif

int main() {
  AnalysisWorkflow workflow("../../../data/Run2012B_SingleMu.root");
  workflow.Run();
  return 0;
}
